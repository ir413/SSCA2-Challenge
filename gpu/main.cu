#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>

#include "Configuration.h"
#include "Graph.h"
#include "ScalableDataGeneration.h"
#include "Timer.h"
#include "Validation.h"


/**
 * Entry point.
 */
int main(int argc, char **argv)
{
  if (argc != 2)
  {
    fprintf(stderr, "Usage: ./SSCA2 <SCALE>\n");
    return EXIT_FAILURE;
  }

  double elapsedTime;

  /* ----------------------------------------- */
  /* Initialization -- Untimed                 */
  /* ----------------------------------------- */
  fprintf(stderr, "Configuring the Benchmark...\n");

  Configuration config;
  configure(atoi(argv[1]), &config);
  fprintf(stderr, "N: %d M: %d\n", config.n, config.m);

  /* ----------------------------------------- */
  /* Scalable Data Generation -- Untimed       */
  /* ------------------------------------------*/
  fprintf(stderr, "Scalable Data Generation...\n");
  elapsedTime = getSeconds();

  // Consturct the tuples structure.
  GraphSDG tuples;
  // Allocate memory required for the tuple arrays.
  tuples.startVertex = (int *) malloc(config.m * sizeof(int));
  tuples.endVertex = (int *) malloc(config.m * sizeof(int));
  tuples.weight = (int *) malloc(config.m * sizeof(int));
  // Allocate memory for the temporary permV array. 
  int *permV = (int *) malloc(config.m * sizeof(int));

  // Consturct the tuples.
  generateScalableData(&config, permV, &tuples);

  // Free the memory used for the temporary permV array.
  free(permV);

  elapsedTime = getSeconds() - elapsedTime;
  fprintf(
      stderr,
      "Time taken for Scalable Data Generation is %9.6lf sec.\n",
      elapsedTime);

  //printTuples(stderr, &tuples);

  /* ----------------------------------------- */
  /* Kernel 1 - Graph Construction             */
  /* ----------------------------------------- */
  fprintf(stderr, "Kernel 1: Constructing the graph...\n");
  elapsedTime = getSeconds();

  // Consturct the graph structure. 
  Graph graph;
  // Allocate memory required for the graph.
  graph.rowOffset = (int *) malloc((config.n + 1) * sizeof(int));
  assert(graph.rowOffset != NULL);
  graph.column = (int *) malloc(config.m * sizeof(int));
  assert(graph.column != NULL);
  graph.weight = (int *) malloc(config.m * sizeof(int)); 
  assert(graph.weight != NULL);

  // Construct the graph.
  constructGraph(&tuples, &graph);

  elapsedTime = getSeconds() - elapsedTime;
  fprintf(stderr, "Time taken for Kernel 1 is %9.6lf sec.\n", elapsedTime);

  //printGraph(stderr, &graph);

  // Clean up the memory used to store generated data.
  free(tuples.weight);
  free(tuples.endVertex);
  free(tuples.startVertex);

  /* ---------------------------------------- */
  /* Kernel 2 - Find max edge weight          */
  /* ---------------------------------------- */
  // TODO

  /* ---------------------------------------- */
  /* Kernel 3 - Graph Extraction              */
  /* ---------------------------------------- */
  // TODO

  /* ---------------------------------------- */
  /* Kernel 4 - Betweenness Centrality        */
  /* ---------------------------------------- */
  fprintf(stderr, "Kernel 4: Computing Betweenness Centrality...\n");
  elapsedTime = getSeconds();

  // TODO
  double *bc = (double *) calloc(config.n, sizeof(double));
  assert(bc != NULL);

  elapsedTime = getSeconds() - elapsedTime;
  fprintf(stderr, "Time taken for Kernel 4 is %9.6lf sec.\n", elapsedTime);

  /* ---------------------------------------- */
  /* Validation                               */
  /* ---------------------------------------- */
  fprintf(stderr, "Validating the results...\n");

  if (isValid(&config, bc))
  {
    fprintf(stderr, "Kernel 4 validation successful!\n");
  }
  else
  {
    fprintf(stderr, "Kernel 4 failed validation!\n");
  }

  // Clean up.
  free(bc);
  free(graph.weight);
  free(graph.column);
  free(graph.rowOffset);
}

